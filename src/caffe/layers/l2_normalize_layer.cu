#include "hip/hip_runtime.h"
#include "caffe/layers/l2_normalize_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "math.h"

namespace caffe {

template <typename Dtype>
void L2NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	for (int i = 0; i < num_; ++i) {
		int start_idx = i * feature_dim_;
		Dtype norm_i;
		caffe_gpu_dot<Dtype>(feature_dim_,
				bottom[0]->cpu_data() + start_idx,
				bottom[0]->cpu_data() + start_idx, &norm_i);
		norm_i = sqrt(norm_i);
		bottom_norm_.mutable_cpu_data()[i] = norm_i;
		caffe_gpu_axpby<Dtype>(feature_dim_, Dtype(1.) / norm_i,
				bottom[0]->gpu_data() + start_idx, Dtype(0.),
				top[0]->mutable_gpu_data() + start_idx);
        }

} 

template <typename Dtype>
void L2NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	for (int i = 0; i < num_; ++i) {
		int start_idx = i * feature_dim_;
		Dtype norm_i = bottom_norm_.cpu_data()[i];
		Dtype norm_i_inv = Dtype(1.) / norm_i;
		Dtype norm_i_tri = pow(norm_i, 3);
		Dtype scal;
		caffe_gpu_dot<Dtype>(feature_dim_,
				bottom[0]->cpu_data() + start_idx,
				top[0]->cpu_diff() + start_idx, &scal);
		scal = -scal / norm_i_tri;
		caffe_gpu_axpby<Dtype>(feature_dim_, norm_i_inv,
				top[0]->gpu_diff() + start_idx, Dtype(0.),
				bottom[0]->mutable_gpu_diff() + start_idx);
                caffe_gpu_axpby<Dtype>(feature_dim_, scal,
				bottom[0]->gpu_data() + start_idx, Dtype(1.),
				bottom[0]->mutable_gpu_diff() + start_idx);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormalizeLayer);

}
